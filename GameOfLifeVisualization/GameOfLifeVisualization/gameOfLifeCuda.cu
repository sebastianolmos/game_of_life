#include "hip/hip_runtime.h"
﻿#include <glad/glad.h>

#include ""
#include <cuda_gl_interop.h>

#include <stdio.h>
#include <assert.h>

#include "lifeKernel.cuh"


extern "C"
{
    void allocateArray(void** devPtr, size_t size)
    {
        hipMalloc(devPtr, size);
    }

    void freeArray(void* devPtr)
    {
        hipFree(devPtr);
    }

    void threadSync()
    {
        hipDeviceSynchronize();
    }

    void copyArrayToDevice(void* device, const void* host, int offset, int size)
    {
        hipMemcpy((char*)device + offset, host, size, hipMemcpyHostToDevice);
    }

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource** cuda_vbo_resource)
    {
        hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, cudaGraphicsMapFlagsNone);
    }

    void unregisterGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        hipGraphicsUnregisterResource(cuda_vbo_resource);
    }

    void* mapGLBufferObject(struct hipGraphicsResource** cuda_vbo_resource)
    {
        void* ptr;
        hipGraphicsMapResources(1, cuda_vbo_resource, 0);
        size_t num_bytes;
        hipGraphicsResourceGetMappedPointer((void**)&ptr, &num_bytes, *cuda_vbo_resource);
        return ptr;
    }

    void unmapGLBufferObject(struct hipGraphicsResource* cuda_vbo_resource)
    {
        hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
    }

    void copyArrayFromDevice(void* host, const void* device,
        struct hipGraphicsResource** cuda_vbo_resource, int size)
    {
        if (cuda_vbo_resource)
        {
            device = mapGLBufferObject(cuda_vbo_resource);
        }

        hipMemcpy(host, device, size, hipMemcpyDeviceToHost);

        if (cuda_vbo_resource)
        {
            unmapGLBufferObject(*cuda_vbo_resource);
        }
    }

    //Round a / b to nearest higher integer value
    uint iDivUp(uint a, uint b)
    {
        return (a % b != 0) ? (a / b + 1) : (a / b);
    }

    // compute grid and thread block size for a given number of elements
    void computeGridSize(uint n, uint blockSize, uint& numBlocks, uint& numThreads)
    {
        numThreads = min(blockSize, n);
        numBlocks = iDivUp(n, numThreads);
    }

    void runGameInDevice(ubyte*& d_data, ubyte*& d_resultData, size_t worldWidth, size_t worldHeight, ushort threads)
    {
        // threads
        size_t calcBlocks = (worldWidth * worldHeight) / threads;
        ushort blocks = (ushort)min((size_t)32768, calcBlocks);

        // set all cells to empty
        //hipMemset(d_resultData, 0x00000000, worldWidth * worldHeight * sizeof(ubyte));
        // execute the kernel
        gameKernel << <blocks, threads >> > ((ubyte*)d_data, (ubyte*)d_resultData, worldWidth, worldHeight);
    }

    /// Runs a kernel for rendering of life world on the screen.
    void runDisplayLifeKernel(const ubyte* d_lifeData, size_t worldWidth, size_t worldHeight, uchar4* destination,
        int destWidth, int destHeight, bool simulateColors) {

        ushort threadsCount = 256;
        assert((worldWidth * worldHeight) % threadsCount == 0);
        size_t reqBlocksCount = (destWidth * destHeight) / threadsCount;
        assert(reqBlocksCount < 65536);
        ushort blocksCount = (ushort)reqBlocksCount;
        displayLifeKernel << <blocksCount, threadsCount >> > ((ubyte*)d_lifeData, uint(worldWidth), uint(worldHeight), (uchar4*)destination,
            destWidth, destHeight, simulateColors);
        hipDeviceSynchronize();
    }

    /// Runs a kernel for cleaning 
    void cleanBufferInDevice(uchar4* buffer, size_t width, size_t height) {

        ushort threadsCount = 256;
        assert((worldWidth * worldHeight) % threadsCount == 0);
        size_t reqBlocksCount = (width * height) / threadsCount;
        assert(reqBlocksCount < 65536);
        ushort blocksCount = (ushort)reqBlocksCount;
        cleanBufferKernel << <blocksCount, threadsCount >> >(  (uchar4*)buffer);
        hipDeviceSynchronize();
    }

}
