﻿
#include "hip/hip_runtime.h"


#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h> 

typedef unsigned char ubyte;
typedef unsigned short ushort;
typedef unsigned int uint;

void initBuffers(ubyte*& data, size_t worldWidth, size_t worldHeight) {
    size_t dataLength = worldWidth * worldHeight;

    for (size_t i = 0; i < dataLength; i++) {
        data[i] = rand() & 1;
    }
}


__global__ void gameKernel(const ubyte* data, ubyte* resultData, uint worldWidth, uint worldHeight)
{
    uint worldSize = worldWidth * worldHeight;
    
    uint cell = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    for (cell; cell < worldSize; cell += blockDim.x * gridDim.x) {
        uint x = cell % worldWidth;
        uint y = cell - x;
        uint xLeft = (x + worldWidth - 1) % worldWidth;
        uint xRight = (x + 1) % worldWidth;
        uint yUp = (y + worldSize - worldWidth) % worldSize;
        uint yDown = (y + worldWidth) % worldSize;

        uint aliveCells = data[xLeft + yUp] + data[x + yUp] + data[xRight + yUp] + data[xLeft + y]
            + data[xRight + y] + data[xLeft + yDown] + data[x + yDown] + data[xRight + yDown];

        resultData[x + y] = aliveCells == 3 || (aliveCells == 2 && data[x + y]) ? 1 : 0;
    }
}

double runGameKernel(ubyte*& d_data, ubyte*& d_resultData, size_t worldWidth,
    size_t worldHeight, size_t iterations, ushort threads) {

    size_t calcBlocks = (worldWidth * worldHeight) / threads;
    ushort blocks = (ushort)std::min((size_t)32768, calcBlocks);
    hipEvent_t start, stop;
    double totalTime = 0.0;

    for (size_t i = 0; i < iterations; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        gameKernel <<<blocks, threads >>> (d_data, d_resultData, worldWidth, worldHeight);
        hipEventRecord(stop);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        double gpuTime = (double)milliseconds / 1000;
        totalTime += gpuTime;

        std::swap(d_data, d_resultData);
    }
    return totalTime;
}

int main(int argc, char* argv[])
{   
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    ubyte* h_data;
    ubyte* h_resultData;

    size_t worldHeight;
    size_t worldWidth;
    size_t dataLength;
    size_t iterations;
    ushort threads;

    if (argc < 5)
    {
        worldWidth = 100;
        worldHeight = 100;
        iterations = 10;
        threads = 32;
    }
    else {
        worldWidth = atoi(argv[1]);
        worldHeight = atoi(argv[2]);
        iterations = atoi(argv[3]);
        threads = atoi(argv[4]);
    }

    dataLength = worldWidth * worldHeight;
    size_t size = dataLength * sizeof(ubyte);

    // Pedir memoria para el host input data
    h_data = new ubyte[dataLength];

    // Pedir memoria para el host output resultData
    h_resultData = new ubyte[dataLength];

    // Verificar si se inicializaron correctamente 
    if (h_data == NULL || h_resultData == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors:\n");
        exit(EXIT_FAILURE);
    }

    /* initialize random seed: */
    srand(time(NULL));
    // Se inicializan los buffers del host
    initBuffers(h_data, worldWidth, worldHeight);

    // Alojar el device input d_data
    ubyte* d_data = NULL;
    err = hipMalloc((void**)&d_data, size);

    // Verificar que el vector data se alojo correctamente en el device
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Alojar el device output d_resultData
    ubyte* d_resultData = NULL;
    err = hipMalloc((void**)&d_resultData, size);

    // Verificar que el vector resultData se alojo correctamente en el device
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector resultData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vector data in host memory to the device input vector in device memory
    err = hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    double time;
    // Launch the game kernel
    time = runGameKernel(d_data, d_resultData, worldWidth, worldHeight, iterations, threads);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch game kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Print the output time
    std::cout << time;

}
